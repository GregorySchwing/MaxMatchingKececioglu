#include "hip/hip_runtime.h"
#include "CSRGraph.cuh"

// kernel function
template <typename T>
__global__ void setNumInArray(T *arrays, T *index, T *value, int num_index)
{
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  if (tid >= num_index || index[tid] < tid)
    return;
  arrays[index[tid]] = value[tid];
}

CSRGraph::CSRGraph(int _n, int _m, int * rows, int * cols) 
{

  m = _m;
  n = _n;
  offsets_h.resize(n + 1);
  degrees_h.resize(n);

  //rows_h.resize(2 * m);
  //cols_h.resize(2 * m);
  rows_h.assign(rows, rows + 2*m);
  cols_h.assign(cols, cols + 2*m);
  vals_h.resize(2 * m, 1);

  rows_d.resize(2 * m);
  // This will be the dst ptr array.
  cols_d.resize(2 * m);
  vals_d.resize(2 * m, 1);

  offsets_d.resize(n + 1);

  keylabel_d.resize(n);
  nonzerodegrees_d.resize(n);
  // This will be the degrees array.
  degrees_d.resize(n);

  mate_h.resize(n,-1);
  mate_d.resize(n,-1);

  createOffsets();
}

void CSRGraph::createOffsets()
  {
    rows_d = rows_h;
    cols_d = cols_h;
    thrust::sort_by_key(thrust::device, rows_d.begin(), rows_d.end(), cols_d.begin());
    thrust::pair<thrust::device_vector<unsigned int>::iterator, thrust::device_vector<unsigned int>::iterator> new_end;
    new_end = thrust::reduce_by_key(thrust::device, rows_d.begin(), rows_d.end(), vals_d.begin(), keylabel_d.begin(), nonzerodegrees_d.begin());
    int block_size = 64;
    int num_blocks = (n + block_size - 1) / block_size;
    unsigned int *degrees_ptr_d = thrust::raw_pointer_cast(degrees_d.data());
    unsigned int *keylabel_ptr_d = thrust::raw_pointer_cast(keylabel_d.data());
    unsigned int *nonzerodegrees_ptr_d = thrust::raw_pointer_cast(nonzerodegrees_d.data());
    setNumInArray<unsigned int><<<num_blocks, block_size>>>(degrees_ptr_d, keylabel_ptr_d, nonzerodegrees_ptr_d, n);
    thrust::inclusive_scan(thrust::device, degrees_d.begin(), degrees_d.end(), offsets_d.begin() + 1); // in-place scan
    offsets_h = offsets_d;
    degrees_h = degrees_d;
    rows_h = rows_d;
    cols_h = cols_d;

    keylabel_d.clear();
    vals_d.clear();
    nonzerodegrees_d.clear();
  }