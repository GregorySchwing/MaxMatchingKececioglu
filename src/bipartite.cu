/*
 * bipartite.c -- Initialize bipartite graph on GPU
 */

/*
 * Copyright 2023 by Greg Schwing
 */


/*
 * Synopsis
 *
 * This implementation of bipartite matching on the GPU can accelerate general
 * graph matching by initializing nearly the entire matching in parallel.
 *
 */

/*
 * Author
 *
 * Greg Schwing
 * gregory.schwing@med.wayne.edu
 *
 * Department of Computer Science
 * Wayne State University
 * Detroit, MI 48201
 *
 */

/*
 * History
 *
 * 22 October 2023
 * First draft.
 *
 */
#include "bipartite.h"
#include "CSRGraph.cuh"
#include "graph.h"

void bipartite(Graph * G){
    int n = G->EL.N;
    int m = G->EL.M;
    CSRGraph csr(n,m);
}